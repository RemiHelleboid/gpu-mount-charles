#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

#include <algorithm>
#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iterator>
#include <random>
#include <string>
#include <thread>

struct point {
    float                              m_x;
    float                              m_y;
    float                              m_z;
    thrust::default_random_engine      m_rng;
    thrust::normal_distribution<float> dist{0, 1};

    point() : m_x{0.0}, m_y{0.0}, m_z{0.0}, m_rng(0.0) {}
    point(uint seed) : m_x{0.0}, m_y{0.0}, m_z{0.0}, m_rng{seed} {}
    point(float x, float y, float z) : m_x{x}, m_y{y}, m_z{z}, m_rng{0} {}
    point(float x, float y, float z, uint seed) : m_x{x}, m_y{y}, m_z{z}, m_rng{seed} {}

    __host__ __device__ void brownian_motion(float dt) {
        m_x += sqrt(dt) * dist(m_rng);
        m_y += sqrt(dt) * dist(m_rng);
        m_z += sqrt(dt) * dist(m_rng);
    }
};

void export_coords(const std::string &filename, const std::vector<point> &vec) {
    std::ofstream my_file(filename);
    my_file << "X,Y,Z" << std::endl;
    for (auto &&my_point : vec) {
        my_file << my_point.m_x << "," << my_point.m_y << "," << my_point.m_z << "\n";
    }
    my_file.close();
}

void export_coords(const std::string &filename, const thrust::host_vector<point> &vec) {
    std::ofstream my_file(filename);
    my_file << "X,Y,Z" << std::endl;
    for (auto &&my_point : vec) {
        my_file << my_point.m_x << "," << my_point.m_y << "," << my_point.m_z << "\n";
    }
    my_file.close();
}

struct move_functor {
    double m_dt;
    move_functor(double dt) : m_dt{dt} {}
    __host__ __device__ point operator()(point &my_point) {
        my_point.brownian_motion(m_dt);
        return my_point;
    }
};

int main(int argc, const char **argv) {
    // generate random data serially
    std::size_t                N         = std::atoi(argv[1]);
    double                     step_time = std::atof(argv[2]);
    thrust::host_vector<point> h_vec(N);
    thrust::host_vector<int>   htmp_vec(N);

    std::generate(h_vec.begin(), h_vec.end(), [n = 1]() mutable {
        uint seed = std::random_device()();
        return point(seed);
    });
    std::generate(htmp_vec.begin(), htmp_vec.end(), [n = 1]() mutable { return n++; });

    // transfer to device and compute sum
    std::cout << "transfer ... " << std::endl;
    thrust::device_vector<point> d_vec = h_vec;
    thrust::device_vector<point> res_vec(d_vec);
    thrust::host_vector<point>   h_res_vec(h_vec.size());
    std::cout << "compute ... " << std::endl;
    move_functor motion{step_time};
    auto         start = std::chrono::high_resolution_clock::now();
    for (std::size_t i = 0; i < 1000; i++) {
        thrust::transform(d_vec.begin(), d_vec.end(), res_vec.begin(), motion);
        // h_vec = d_vec;
        // export_coords("traj/point_coords." + std::to_string(i) + ".csv", h_vec);
    }

    auto stop     = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start);
    std::cout << std::scientific << "Time of simulation GPU : " << duration.count() << " ns" << std::endl;

    auto start_serial = std::chrono::high_resolution_clock::now();
    for (std::size_t i = 0; i < 1000; i++) {
        std::transform(h_vec.begin(), h_vec.end(), h_res_vec.begin(), motion);
    }
    auto stop_serial     = std::chrono::high_resolution_clock::now();
    auto duration_serial = std::chrono::duration_cast<std::chrono::nanoseconds>(stop_serial - start_serial);
    std::cout << std::scientific << "Time of simulation CPU : " << duration_serial.count() << " ns" << std::endl;
    std::cout << std::scientific << "Time CPU / Time GPU = " << 1.0 * duration_serial / duration << std::endl;

    h_vec = d_vec;

    return 0;
}